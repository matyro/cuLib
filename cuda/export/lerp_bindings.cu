#include "hip/hip_runtime.h"
#include <pybind11/stl.h>
#include <pybind11/pybind11.h>
//#include <pybind11/numpy.h>
namespace py = pybind11;

#include "cuLib/texture.cuh"
#include "cuLib/memory.cuh"
#include "cuLib/context.cuh"

__global__ void lerp_kernel(hipTextureObject_t tex, float* io, const unsigned int N)
{int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= N)
        return;

    const float tex_coords = (float)io[idx] + 0.5;  
    io[idx] = tex2D<float>(tex, tex_coords, 1 );    
}

std::vector<double> execute_lerp(std::vector<double> input, std::vector<float> table, double scale)
{       
    hipError_t last_error_;

    Memory<float> io(input.size());

    for (int i = 0; i < input.size(); i++)
    {       
        io[i] = input[i];
    }
    io.copyToDevice();

    hipArray *dArray_;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    last_error_ = hipMallocArray(&dArray_, &channelDesc, table.size(), 1, hipArrayDefault);
    if (last_error_ != hipSuccess)
        throw std::runtime_error("hipMallocArray failed with " + std::string(hipGetErrorString(last_error_)));


    last_error_ = hipMemcpy2DToArray(dArray_, 0, 0, table.data(), table.size() * sizeof(float), table.size() * sizeof(float), 1, hipMemcpyHostToDevice);
    if (last_error_ != hipSuccess)
        throw std::runtime_error("hipMemcpy2DToArray failed with " + std::string(hipGetErrorString(last_error_)) + " " + std::to_string(last_error_));

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = dArray_;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType; // Read data as provided type, no casting
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.normalizedCoords = 0;

    hipTextureObject_t textureObject_;

    // create texture object: we only have to do this once!
    last_error_ = hipCreateTextureObject(&textureObject_, &resDesc, &texDesc, NULL);
    if (last_error_ != hipSuccess)
        throw std::runtime_error("hipCreateTextureObject failed with " + std::string(hipGetErrorString(last_error_)));

    lerp_kernel<<<(input.size() + 255) / 256, 256>>>(textureObject_, io.getDevicePtr(), input.size() );

    auto error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        std::cout << "Kernel failed: " << hipGetErrorString(error) << std::endl;
        throw std::runtime_error("Kernel failed");
    }

    io.copyToHost();

    return std::vector<double>(io.begin(), io.end());
}


/*
std::vector<double> execute_lerp(std::vector<double> input, std::vector<float> table, double scale)
{
    Context cudaContext(0);  
    cudaContext.synchronize(); 

    Memory<float> io(input.size());
    Texture tex(table.size());

    tex.copyToTexture(table.data(), table.size());


    lerp_kernel<<<(input.size() + 255) / 256, 256>>>(tex.getTextureObject(), io.getDevicePtr(), input.size() );

    // Wait until kernel is finished
    auto error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        std::cout << "Kernel failed: " << hipGetErrorString(error) << std::endl;
        throw std::runtime_error("Kernel failed " + std::string(hipGetErrorString(error)) + " " + std::to_string(error));
    }
    
  
    io.copyToHost();
  
    std::vector<double> result(input.size());

    std::copy(io.begin(), io.end(), result.begin());
    return result;
}*/

/*
py::array_t<double> execute_lerp_numpy(py::array_t<float, py::array::c_style | py::array::forcecast> tablePy, double scale, py::array_t<double, py::array::c_style | py::array::forcecast> inputPy)
{
    Context cudaContext(0);

    py::buffer_info table = tablePy.request();
    unsigned int Ntable = table.size;

    py::buffer_info input = inputPy.request();
    unsigned int Ninput = input.size;

    std::cout << "Tex Size: " << Ntable << " Querry Size:" << Ninput << std::endl;

    Texture tex(Ntable);
    tex.copyToTexture(reinterpret_cast<float*>(table.ptr), Ntable);



    Memory<float> io(Ninput);
    for (int i = 0; i < Ninput; i++)
    {
        io[i] = reinterpret_cast<double*>(input.ptr)[i];
    }
    io.copyToDevice();

    cudaContext.synchronize();
   
    std::cout << "Kernel: " << std::endl;
    lerp<<<(Ninput + 255) / 256, 256>>>(tex.getTextureObject(), io.getDevicePtr(), scale, Ninput);

    // Wait until kernel is finished
    auto error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        std::cout << "Kernel failed: " << hipGetErrorString(error) << std::endl;
        throw std::runtime_error("Kernel failed");
    }
    
    std::cout << "Kernel finished" << std::endl;

    //io.copyToHost();

    std::cout << "Copy finished" << std::endl;

    auto result = py::array_t<double>(input.size);
    py::buffer_info output = result.request();

    std::copy(io.begin(), io.end(), reinterpret_cast<double*>(output.ptr));

    std::cout << "result: " << std::endl;
    for(int i = 0; i< Ninput; i++)
    {
        std::cout << reinterpret_cast<double*>(output.ptr)[i] << std::endl;
    }

    return result;
}*/

void bind_lerp(py::module &m)
{
   // m.def("execute_lerp_numpy", &execute_lerp, "execute lerp");
    m.def("execute_lerp", &execute_lerp, "execute lerp");
}