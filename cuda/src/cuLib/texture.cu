
#include <memory>

#include <stdexcept>

#include <iostream>

#include "cuLib/texture.cuh"

__host__ Texture::Texture(const size_t N) : width_(N)
{   

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    last_error_ = hipMallocArray(&dArray_, &channelDesc, sizeof(float) * width_, 1, hipArrayDefault);
    if (last_error_ != hipSuccess)
        throw std::runtime_error("Texture::Texture: hipMallocArray failed width " + std::string(hipGetErrorString(last_error_)));
}

__host__ Texture::~Texture()
{
    hipFreeArray(dArray_);

    if (textureObject_ != 0)
        hipDestroyTextureObject(textureObject_);
}

__host__ void Texture::copyToTexture(const float *const data, const unsigned int N)
{
    if (N != width_)
        throw std::runtime_error("Texture::copyToTexture: data size does not match texture size");

    // (hipArray_t dst, size_t wOffset, size_t hOffset, const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind);
    last_error_ = hipMemcpy2DToArray(dArray_, 0, 0, data, N * sizeof(float), N * sizeof(float), 1, hipMemcpyHostToDevice);
    //last_error_ = hipMemcpyToArray(dArray_, 0, 0, data, width_ * sizeof(float), hipMemcpyHostToDevice);
    if (last_error_ != hipSuccess)
        throw std::runtime_error("Texture::copyToTexture: hipMemcpyToArray failed width " + std::string(hipGetErrorString(last_error_)));
}

__host__ hipTextureObject_t Texture::getTextureObject()
{
    if (textureObject_ == 0)
    {
        // create texture object
        memset(&resDesc_, 0, sizeof(resDesc_));
        resDesc_.resType = hipResourceTypeArray;
        resDesc_.res.array.array = dArray_;

        memset(&texDesc_, 0, sizeof(texDesc_));
        texDesc_.readMode = hipReadModeElementType; // Read data as provided type, no casting
        texDesc_.filterMode = hipFilterModeLinear;
        texDesc_.addressMode[0] = hipAddressModeWrap;
        texDesc_.normalizedCoords = 0;

        hipResourceViewDesc viewDesc;
        memset(&viewDesc, 0, sizeof(viewDesc));
        viewDesc.format = hipResViewFormatFloat1;
        viewDesc.width = width_;
        viewDesc.height = 1;

        // create texture object: we only have to do this once!
        last_error_ = hipCreateTextureObject(&textureObject_, &resDesc_, &texDesc_, NULL);
        if (last_error_ != hipSuccess)
            throw std::runtime_error("Texture::getTextureObject: hipCreateTextureObject failed with " + std::string(hipGetErrorString(last_error_)));
    }
    return textureObject_;
}
