

#include <iostream>
#include <stdexcept>

template <class TType>
Memory<TType>::Memory(unsigned int size) : hData_ptr_(nullptr), dData_ptr_(nullptr), size_(size)
{

    last_error_= hipHostMalloc(&hData_ptr_, size_ * sizeof(TType));
    if (last_error_!= hipSuccess)
    {
        std::cout << "error allocating pinned host memory: " << hipGetErrorString(last_error_) << std::endl;
        exit(-1);
    }

    last_error_= hipMalloc(&dData_ptr_, size_ * sizeof(TType));
    if (last_error_!= hipSuccess)
    {
        std::cout << "error allocating device memory: " << hipGetErrorString(last_error_) << std::endl;
        exit(-1);
    }
}

template <class TType>
Memory<TType>::~Memory()
{
    hipHostFree(hData_ptr_);
    hipFree(dData_ptr_);   
}

template <class TType>
void Memory<TType>::copyToDevice()
{
    last_error_= hipMemcpy(dData_ptr_, hData_ptr_, size_ * sizeof(TType), hipMemcpyHostToDevice);
    if (last_error_!= hipSuccess)
    {
        std::cout << "error copying data to device: " << hipGetErrorString(last_error_) << std::endl;
        exit(-1);
    }
}

template <class TType>
void Memory<TType>::copyToHost()
{  
    last_error_= hipMemcpy(hData_ptr_, dData_ptr_, size_ * sizeof(TType), hipMemcpyDeviceToHost);
    if (last_error_!= hipSuccess)
    {
        std::cout << "error copying data to host: " << hipGetErrorString(last_error_) << std::endl;
        exit(-1);
    }
}

template <class TType>
TType* Memory<TType>::getHostPtr()
{
    return hData_ptr_;
}

template <class TType>
TType* Memory<TType>::getDevicePtr()
{
    return dData_ptr_;
}

template <class TType>
TType *Memory<TType>::begin()
{
    return hData_ptr_;
}

template <class TType>
TType *Memory<TType>::end()
{
    return hData_ptr_ + size_;
}

template <class TType>
const TType *Memory<TType>::cbegin() const
{
    return hData_ptr_;
}

template <class TType>
const TType *Memory<TType>::cend() const
{
    return hData_ptr_ + size_;
}

template <class TType>
TType &Memory<TType>::operator[](unsigned int index)
{
    if (index >= size_)
        throw std::out_of_range("Index out of range");

    return hData_ptr_[index];
}

template <class TType>
const TType &Memory<TType>::operator[](unsigned int index) const
{
    if (index >= size_)
        throw std::out_of_range("Index out of range");

    return hData_ptr_[index];
}